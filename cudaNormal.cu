//%%cu
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

void inicializar_matriz(int n, float *matriz) {
    for (int i = 0; i < n * n; i++) {
        matriz[i] = rand() % 10;
    }
}

void imprimir_matriz(int n, float *matriz) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            printf("%.2f ", matriz[i * n + j]);
        }
        printf("\n");
    }
}

void guardar_matriz_txt(int n, float *matriz, const char *nombre_archivo) {
    FILE *archivo = fopen(nombre_archivo, "w");
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            fprintf(archivo, "%.2f ", matriz[i * n + j]);
        }
        fprintf(archivo, "\n");
    }
    fclose(archivo);
}

__global__ void matriz_multiplicar(float *a, float *b, float *c, int n) {
    int fila = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float suma = 0;
    if (fila < n && col < n) {
        for (int i = 0; i < n; i++) {
            suma += a[fila * n + i] * b[i * n + col];
        }
        c[fila * n + col] = suma;
    }
}

int main() {

    int n;
    printf("Ingrese el tamaño de la matriz (n): ");
    scanf("%d", &n);
    int BLOCKS_PER_GRID;
    printf("Ingrese el número de bloques por cluster (BLOCKS_PER_GRID): ");
    scanf("%d", &n);
    int THREADS_PER_BLOCK;
    printf("Ingrese el número de hilos por bloque (THREADS_PER_BLOCK): ");
    scanf("%d", &n);
    
    // Asignar memoria en el host
    float *matriz_a = (float *) malloc(sizeof(float) * n * n);
    float *matriz_b = (float *) malloc(sizeof(float) * n * n);
    float *matriz_res = (float *) malloc(sizeof(float) * n * n);

    // Inicializar matrices
    inicializar_matriz(n, matriz_a);
    inicializar_matriz(n, matriz_b);

    // Imprimir matrices originales
    //printf("Matriz A:\n");
    //imprimir_matriz(n, matriz_a);
    //printf("Matriz B:\n");
    //imprimir_matriz(n, matriz_b);

    // Asignar memoria en el dispositivo
    float *d_a, *d_b, *d_res;
    hipMalloc((void **) &d_a, sizeof(float) * n * n);
    hipMalloc((void **) &d_b, sizeof(float) * n * n);
    hipMalloc((void **) &d_res, sizeof(float) * n * n);

    // Copiar matrices del host al dispositivo
    hipMemcpy(d_a, matriz_a, sizeof(float) * n * n, hipMemcpyHostToDevice);
    hipMemcpy(d_b, matriz_b, sizeof(float) * n * n, hipMemcpyHostToDevice);

    // Definir la cantidad de threads 
    dim3 block_size(BLOCKS_PER_GRID, BLOCKS_PER_GRID, 1);
    dim3 grid_size(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);

    // Lanzar kernel para multiplicar matrices 
    matriz_multiplicar<<<grid_size, block_size>>>(d_a, d_b, d_res, n);

    // Copiar matriz resultado del dispositivo al host
    hipMemcpy(matriz_res, d_res, sizeof(float) * n * n, hipMemcpyDeviceToHost);

    // Imprimir matriz resultado
    //printf("Matriz resultado:\n");
    //imprimir_matriz(n, matriz_res);
    guardar_matriz_txt(n,matriz_a, "matrizA.txt");
    guardar_matriz_txt(n,matriz_b, "matrizB.txt");
    guardar_matriz_txt(n,matriz_res, "matrizC.txt");

    // Liberar memoria
    free(matriz_a);
    free(matriz_b);
    free(matriz_res);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_res);

    return 0;
}